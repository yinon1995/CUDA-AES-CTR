﻿// System includes
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <ctime>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA

#include <>
#include <hip/device_functions.h>

// Custom header 
#include "kernel.h"
//
#include "128-ctr.cuh"



int main() {

	// AES-128 Exhaustive Search
	main128Ctr();


	return 0;
}
